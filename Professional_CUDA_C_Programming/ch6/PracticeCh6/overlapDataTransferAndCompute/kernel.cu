#include <stdio.h>
#include <hip/hip_runtime.h>

#include <cmath>
#include <cstring>

#define NSTREAM 4
#define BDIM 128

void initialData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{
		ip[i] = (float)(rand() & 0xFF) / 10.0f;
	}
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
	for (int idx = 0; idx < N; idx++)
		C[idx] = A[idx] + B[idx];
}

__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		for (int i = 0; i < N; ++i)
		{
			C[idx] = A[idx] + B[idx];
		}
	}
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
	double epsilon = 1.0E-8;
	bool match = true;

	for (int i = 0; i < N; i++)
	{
		if (abs(hostRef[i] - gpuRef[i]) > epsilon)
		{
			match = false;
			printf("Arrays do not match!\n");
			printf("host %5.2f gpu %5.2f at %d\n", hostRef[i], gpuRef[i], i);
			break;
		}
	}

	if (match)
		printf("Arrays match.\n\n");
}

int main(int argc, char **argv)
{
	printf("> %s Starting...\n", argv[0]);

	int dev = 0;
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	printf("> Using Device %d: %s\n", dev, deviceProp.name);
	hipSetDevice(dev);

	// check if device support hyper-q
	if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5))
	{
		if (deviceProp.concurrentKernels == 0)
		{
			printf("> GPU does not support concurrent kernel execution (SM 3.5 or higher required)\n");
			printf("> CUDA kernel runs will be serialized\n");
		}
		else
		{
			printf("> GPU does not support HyperQ\n");
			printf("> CUDA kernel runs will have limited concurrency\n");
		}
	}

	printf("> Compute Capability %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

	// set up data size of vectors
	int nElem = 1 << 16;
	printf("> vector size = %d\n", nElem);
	size_t nBytes = nElem * sizeof(float);

	// malloc pinned host memory for async memcpy
	float *h_A, *h_B, *hostRef, *gpuRef;
	hipHostAlloc((void**)&h_A, nBytes, hipHostMallocDefault);
	hipHostAlloc((void**)&h_B, nBytes, hipHostMallocDefault);
	hipHostAlloc((void**)&gpuRef, nBytes, hipHostMallocDefault);
	hipHostAlloc((void**)&hostRef, nBytes, hipHostMallocDefault);

	// initialize data at host side
	initialData(h_A, nElem);
	initialData(h_B, nElem);
	memset(hostRef, 0, nBytes);
	memset(gpuRef, 0, nBytes);

	// add vector at host side for result checks
	sumArraysOnHost(h_A, h_B, hostRef, nElem);

	// malloc device global memory
	float *d_A, *d_B, *d_C;
	hipMalloc((float**)&d_A, nBytes);
	hipMalloc((float**)&d_B, nBytes);
	hipMalloc((float**)&d_C, nBytes);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// invoke kernel at host side
	dim3 block(BDIM);
	dim3 grid((nElem + block.x - 1) / block.x);
	printf("> grid (%d, %d) block (%d, %d)\n", grid.x, grid.y, block.x,block.y);

	// sequential operation
	hipEventRecord(start, 0);
	hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float memcpy_h2d_time;
	hipEventElapsedTime(&memcpy_h2d_time, start, stop);

	hipEventRecord(start, 0);
	sumArrays <<<grid, block >>>(d_A, d_B, d_C, nElem);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float kernel_time;
	hipEventElapsedTime(&kernel_time, start, stop);
	hipEventRecord(start, 0);
	hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float memcpy_d2h_time;
	hipEventElapsedTime(&memcpy_d2h_time, start, stop);
	float itotal = kernel_time + memcpy_h2d_time + memcpy_d2h_time;

	printf("\n");
	printf("Measured timings (throughput):\n");
	printf(" Memcpy host to device\t: %f ms (%f GB/s)\n", memcpy_h2d_time, (nBytes * 1e-6) / memcpy_h2d_time);
	printf(" Memcpy device to host\t: %f ms (%f GB/s)\n", memcpy_d2h_time, (nBytes * 1e-6) / memcpy_d2h_time);
	printf(" Kernel\t\t\t: %f ms (%f GB/s)\n", kernel_time, (nBytes * 2e-6) / kernel_time);
	printf(" Total\t\t\t: %f ms (%f GB/s)\n", itotal, (nBytes * 2e-6) / itotal);

	// grid parallel operation
	int iElem = nElem / NSTREAM;
	size_t iBytes = iElem * sizeof(float);
	grid.x = (iElem + block.x - 1) / block.x;

	hipStream_t stream[NSTREAM];

	for (int i = 0; i < NSTREAM; ++i)
	{
		hipStreamCreate(&stream[i]);
	}

	hipEventRecord(start, 0);

	// initiate all work on the device asynchronously in depth-first order
	for (int i = 0; i < NSTREAM; ++i)
	{
		int ioffset = i * iElem;
		hipMemcpyAsync(&d_A[ioffset], &h_A[ioffset], iBytes, hipMemcpyHostToDevice, stream[i]);
		hipMemcpyAsync(&d_B[ioffset], &h_B[ioffset], iBytes, hipMemcpyHostToDevice, stream[i]);
		sumArrays <<<grid, block, 0, stream[i] >>>(&d_A[ioffset], &d_B[ioffset], &d_C[ioffset], iElem);
		hipMemcpyAsync(&gpuRef[ioffset], &d_C[ioffset], iBytes, hipMemcpyDeviceToHost, stream[i]);
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float execution_time;
	hipEventElapsedTime(&execution_time, start, stop);

	printf("\n");
	printf("Actual results from overlapped data transfers:\n");
	printf(" overlap with %d streams : %f ms (%f GB/s)\n", NSTREAM, execution_time, (nBytes * 2e-6) / execution_time);
	printf(" speedup                : %f \n", ((itotal - execution_time) * 100.0f) / itotal);

	// check kernel error
	hipGetLastError();

	// check device results
	checkResult(hostRef, gpuRef, nElem);

	// free device global memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// free host memory
	hipHostFree(h_A);
	hipHostFree(h_B);
	hipHostFree(hostRef);
	hipHostFree(gpuRef);

	// destroy events
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// destroy streams
	for (int i = 0; i < NSTREAM; ++i)
	{
		hipStreamDestroy(stream[i]);
	}

	hipDeviceReset();

	system("Pause");
	return(0);
}