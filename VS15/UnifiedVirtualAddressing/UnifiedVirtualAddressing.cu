
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <ctime>


__global__ void sumArraysZeroCopyUVA(float* A, float* B, float* C, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N)
		C[idx] = A[idx] + B[idx];
}

void sumArraysOnHost(float* A, float* B, float* C, int N)
{
	for (int i = 0; i<N; ++i)
	{
		C[i] = A[i] + B[i];
	}
}

void initialData(float* ip, int size)
{
	time_t t;
	srand((unsigned int)time(&t));
	for (int i = 0; i<size; ++i)
	{
		ip[i] = (float)(rand() & 0xFF) / 10.0f;
	}
}

void checkResult(float* hostResult, float* deviceResult, const int N)
{
	double epsilon = 1.0E-8;
	int match = 1;
	for (int i = 0; i<N; ++i)
	{
		if (abs(hostResult[i] - deviceResult[i]) > epsilon)
		{
			match = 0;
			printf("Array do not match\n");
			printf("host %5.2f gpu %5.2f at current %d\n", hostResult[i], deviceResult[i], i);
			break;
		}
	}
	if (match)
		printf("Array match\n");

	return;
}

int main(int argc, char* argv[])
{
	int dev = 0;
	hipSetDevice(dev);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

	if (deviceProp.canMapHostMemory == false)
	{
		printf("Device %d dose not support mapping CPU host memory!\n", dev);
		hipDeviceReset();
		return -1;
	}

	printf("Using Device %d, %s\n", dev, deviceProp.name);

	int iPower = 10;
	if (argc > 1)
	{
		iPower = atoi(argv[1]);
	}
	int nElem = 1 << iPower;
	size_t nBytes = nElem * sizeof(float);

	// part 1 use device memory
	float* h_a;
	float* h_b;
	float* d_c;
	float* hostRef;
	float* gpuRef;

	unsigned int flags = hipHostMallocMapped;
	hipHostAlloc((void**)&h_a, nBytes, flags);
	hipHostAlloc((void**)&h_b, nBytes, flags);
	hipHostAlloc((void**)&d_c, nBytes, flags);

	hostRef = (float*)malloc(nBytes);

	initialData(h_a, nElem);
	initialData(h_b, nElem);
	memset(hostRef, 0, nBytes);

	sumArraysOnHost(h_a, h_b, hostRef, nElem);

	int nLen = 32;
	dim3 block(nLen);
	dim3 grid((nElem + block.x - 1) / block.x);

	sumArraysZeroCopyUVA<<<grid, block >>>(h_a, h_b, d_c, nElem);

	checkResult(hostRef, d_c, nElem);

	hipHostFree(d_c);
	hipHostFree(h_a);
	hipHostFree(h_b);

	free(hostRef);

	hipDeviceReset();

	system("Pause");
	return 0;
}
