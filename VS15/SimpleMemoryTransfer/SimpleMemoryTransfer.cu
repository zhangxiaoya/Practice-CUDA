
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

int main(int argc, char* argv[])
{
	int dev = 0;
	hipSetDevice(dev);
	unsigned int isize = 1 << 22;
	unsigned int nbytes = isize * sizeof(float);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	printf("%s starting at ", argv[0]);
	printf("device %d: %s memory size %d nbyte %5.2f MB\n", dev, deviceProp.name, isize, nbytes/(1024.0f * 1024.0f));

	float *h_a = (float*)malloc(nbytes);
	float *d_a;
	hipMalloc((float**)&d_a, nbytes);

	for(unsigned int i = 0; i< isize; ++i)
	{
		h_a[i] = 0.5f;
	}

	hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost);

	hipFree(d_a);
	free(h_a);

	hipDeviceReset();
	system("Pause");
    return 0;
}
