
#include "hip/hip_runtime.h"

#include <iostream>

#include <stdio.h>

__device__ float devData;

__global__ void checkGlobalVariable()
{
	printf("Device: The value of global variable is %f\n", devData);
	devData += 2.0;
}

int main()
{
	float value = 3.14f;
	hipMemcpyToSymbol(HIP_SYMBOL(&devData), &value, sizeof(value));
	printf("Host: copied %f to the global variable\n", value);

	checkGlobalVariable<<<1, 1 >>> ();

	hipMemcpyFromSymbol(&value, HIP_SYMBOL(&devData), sizeof(value));
	printf("Host: Value changed by kernel to %f\n", value);

	hipDeviceReset();

	system("Pause");
    return 0;
}