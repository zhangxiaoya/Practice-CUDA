#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define CHECK(call)                                                         \
{                                                                           \
    const hipError_t error = call;                                         \
    if(error != hipSuccess)                                                \
    {                                                                       \
        printf("Error: %s:%d, ", __FILE__, __LINE);                         \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1);                                                            \
    }                                                                       \
}                                                                           \

double cpuSecond()
{
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

__global__ void sumArraysOnDevice(float* A, float* B, float* C)
{
    C[threadIdx.x] = A[threadIdx.x] + B[threadIdx.x];
}

void sumArraysOnHost(float* A, float* B, float* C, int N)
{
    for(int i=0;i<N;++i)
    {
        C[i] = A[i] + B[i];
    }
}

void initialData(float* ip, int size)
{
    time_t t;
    srand((unsigned int) time(&t));
    for(int i=0;i<size;++i)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

void checkResult(float* hostResult, float* deviceResult, const int N)
{
    double epsilon = 1.0E-8;
    int match = 1;
    for(int i =0;i<N;++i)
    {
        if(abs(hostResult[i] - deviceResult[i]) > epsilon)
        {
            match = 0;
            printf("Array do not match\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostResult[i], deviceResult[i], i);
            break;
        }
    }
    if(match)
        printf("Array match\n");

    return;
}

int main(void)
{
    // int nElem = 1024;
    int nElem = 1 << 10;
    size_t nBytes = nElem * sizeof(float);

    float *A, *B, *C;
    A = (float*)malloc(nBytes);
    B = (float*)malloc(nBytes);
    C = (float*)malloc(nBytes);

    float *dA, *dB, *dC;
    float *gpuResult;
    hipMalloc((float**)&dA, nBytes);
    hipMalloc((float**)&dB, nBytes);
    hipMalloc((float**)&dC, nBytes);
    gpuResult = (float*)malloc(nBytes);

    initialData(A, nElem);
    initialData(B, nElem);

    hipMemcpy(dA,A,nBytes,hipMemcpyHostToDevice);
    hipMemcpy(dB,B,nBytes,hipMemcpyHostToDevice);

    dim3 block(nElem);
    dim3 grid(nElem / block.x);

    double iStart = cpuSecond();
    sumArraysOnDevice<<<grid,block>>>(dA,dB,dC);
    hipDeviceSynchronize();
    double iElaps = cpuSecond() - iStart;

    printf("GPU time is %f\n",iElaps);

    hipMemcpy(gpuResult,dC,nBytes,hipMemcpyDeviceToHost);

    sumArraysOnHost(A,B,C,nElem);

    checkResult(C,gpuResult,nElem);
    free(A);
    free(B);
    free(C);
    free(gpuResult);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return 0;
}