#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>

#define CHECK(call)                                                          \
{                                                                            \
    const hipError_t error = call;                                          \
    if(error 1= hipSuccess)                                                 \
    {                                                                        \
        printf("Error: %s : %d, ", __FILE__, __LINE__);                      \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                             \
    }                                                                        \
}

void printMatrix(int* data, const int nx, const int ny)
{
    int* pdata = data;
    printf("\nMatrix : (%d, %d)\n", nx, ny);
    for(int i=0;i<ny;++i)
    {
        for(int j =0;j<nx;++j)
        {
            printf("%3d", pdata[j]);
        }
        pdata += nx;
        printf("\n");
    }
    printf("\n");
}

__global__ void PrintThreadBlockIndex(int* data, const int nx, const int ny)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idex = ix + iy * nx;

    printf("thread index (%d, %d), block index (%d, %d) coordinate (%d, %d), global index %d and value %d\n", 
            threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ix, iy,idex, data[idex]);
}

void InitData(int* data, const int size)
{
    time_t t;
    srand((unsigned)time(&t));
    for(int i =0; i < size; ++i)
    {
        data[i] = (int)(rand() & 0xFF) /10.0f;
    }
}

int main(int argc, char const *argv[])
{
    printf("%s Starting ... \n", argv[0]);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,dev);
    printf("Using Device %d, %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    int nx = 8;
    int ny = 6;
    int nxy = nx * ny;

    size_t nBytes = nxy * sizeof(int);

    int* hA;
    hA = (int*)malloc(nBytes);

    InitData(hA, nxy);
    printMatrix(hA, nx, ny);

    int* dMat;
    hipMalloc((void**)&dMat, nBytes);

    hipMemcpy(dMat, hA, nBytes, hipMemcpyHostToDevice);

    dim3 block(4,2);
    dim3 grid((nx + block.x -1) / block.x, (ny + block.y - 1) / block.y);

    PrintThreadBlockIndex<<<grid, block>>>(dMat, nx, ny);

    hipDeviceSynchronize();
    hipFree(dMat);
    free(hA);

    hipDeviceReset();
    return 0;
}