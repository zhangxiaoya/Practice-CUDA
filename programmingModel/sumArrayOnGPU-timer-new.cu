#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>

#define CHECK(call)                                                          \
{                                                                            \
    const hipError_t error = call;                                          \
    if(error 1= hipSuccess)                                                 \
    {                                                                        \
        printf("Error: %s : %d, ", __FILE__, __LINE__);                      \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                             \
    }                                                                        \
}

void CheckResult(float* hostResult, float* deviceResult, const int N)
{
    double epsilon = 1.0E-8;
    int match = 1;
    for(int i = 0; i < N; ++i)
    {
        if(abs(hostResult[i] - deviceResult[i]) > epsilon)
        {
            match = 0;
            printf("Array do not match!\n");
            printf("Host %5.2gf GPU %5.2f at current %d \n", hostResult[i], deviceResult[i], i);
            break;
        }
    }
    if(match == 1)
    {
        printf("Array match.\n\n");
    }
    return;
}

void InitData(float* data, const int size)
{
    time_t t;
    srand((unsigned)time(&t));
    for(int i =0; i < size; ++i)
    {
        data[i] = (float)(rand() & 0xFF) /10.0f;
    }
}

void SumArrayOnHost(float* A, float* B, float* C, const int size)
{
    for(int i =0; i< size;++i)
    {
        C[i] = A[i] + B[i];
    }
}

__global__ void SumArrayOnDevice(float* A, float* B, float* C, const int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < size)
    {
        C[i] = A[i] + B[i];
    }
}

double cpuSecond()
{
    timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

int main(int argc, char const *argv[])
{
    printf("%s Starting ... \n", argv[0]);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,dev);
    printf("Using Device %d, %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    int nElem = 1 << 24;
    printf("Array size is %d\n", nElem);

    size_t nBytes = nElem * sizeof(float);

    float* hA;
    float* hB;
    float* hostResult;
    float* deviceResult;
    hA = (float*)malloc(nBytes);
    hB = (float*)malloc(nBytes);
    hostResult = (float*)malloc(nBytes);
    deviceResult = (float*)malloc(nBytes);

    InitData(hA, nElem);
    InitData(hB, nElem);

    memset(hostResult, 0, nBytes);
    memset(deviceResult, 0, nBytes);

    float* dA;
    float* dB;
    float* dC;
    hipMalloc((float**)&dA, nBytes);
    hipMalloc((float**)&dB, nBytes);
    hipMalloc((float**)&dC, nBytes);

    hipMemcpy(dA, hA, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, nBytes, hipMemcpyHostToDevice);

    double iStart, iElaps;

    int iLen = 1024;
    dim3 block(iLen);
    dim3 grid((nElem + block.x - 1) / block.x);

    iStart = cpuSecond();
    SumArrayOnDevice<<<grid,block>>>(dA, dB, dC, nElem);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    printf("SumArrayOnGPU <<<%d, %d>>> time elapsed %f sec\n", grid.x, block.x, iElaps);

    hipMemcpy(deviceResult, dC, nBytes, hipMemcpyDeviceToHost);

    iStart = cpuSecond();
    SumArrayOnHost(hA, hB,hostResult,nElem);
    iElaps = cpuSecond() - iStart;
    printf("SumArrayOnCPU time elapsed %f sec\n", iElaps);

    CheckResult(hostResult, deviceResult,nElem);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    free(hA);
    free(hB);
    free(hostResult);
    free(deviceResult);

    return 0;
}