#include <stdio.h>
#include <hip/hip_runtime.h>

int main()
{
    int nElem = 1024;

    // define grid and block structure
    dim3 block(1024);
    dim3 grid((nElem + block.x -1) / block.x);
    printf("grid.x %d block.x %d\n", grid.x, block.x);

    // reset
    block.x = 512;
    grid.x = (nElem + block.x - 1) / block.x;
    printf("grid.x %d block.x %d\n", grid.x, block.x);

    // reset
    block.x = 256;
    grid.x = (nElem + block.x - 1) / block.x;
    printf("grid.x %d block.x %d\n", grid.x, block.x);

    // reset
    block.x = 128;
    grid.x = (nElem + block.x - 1) / block.x;
    printf("grid.x %d block.x %d\n", grid.x, block.x);

    hipDeviceReset();
    return 0;
}