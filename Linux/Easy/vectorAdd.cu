#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void VectorAdd(int* A, int* B, int* C, int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx > N )
        return;
    C[idx] = A[idx] + B[idx];
}

void GenerateData(int* data, int N)
{
    for(int i = 0; i< N; ++i)
        (*(data+i)) = i;
}
void VectorAddOnDevice(int N)
{
    int* HA = (int*)malloc(sizeof(int) * N);
    int* HB = (int*)malloc(sizeof(int) * N);
    int* HC = (int*)malloc(sizeof(int) * N);

    GenerateData(HA, N);
    GenerateData(HB, N);

    int nBytes = N * sizeof(int);
    int* DA;
    int* DB;
    int* DC;
    hipMalloc((int**)&DA, nBytes);
    hipMalloc((int**)&DB, nBytes);
    hipMalloc((int**)&DC, nBytes);

    hipMemcpy(DA, HA,nBytes, hipMemcpyHostToDevice);
    hipMemcpy(DB, HB,nBytes, hipMemcpyHostToDevice);

    dim3 block(32);
    dim3 grid((N + block.x -1) / block.x);

    VectorAdd<<<grid, block>>>(DA, DB,DC,N);
    hipMemcpy(HC, DC, nBytes, hipMemcpyDeviceToHost);

    for(int i = 0; i < N;++i)
        printf("%d ", HC[i]);
    printf("\n");

    hipFree(DA);
    hipFree(DB);
    hipFree(DC);

    free(HA);
    free(HB);
    free(HC);
}