
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU()
{
    printf("Hello World From GPU!\n");
}

int main()
{
    // helloFromGPU
    printf("Hello World From CPU!\n");

    // helloFromGPU
    helloFromGPU<<<1, 10>>>();
    hipDeviceReset();
    return 0;
}